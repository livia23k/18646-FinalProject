#include "hip/hip_runtime.h"
//
// smallpt, a Path Tracer by Kevin Beason, 2008
// Make : g++ -O3 smallpt_serial.cpp -o smallpt_serial
//        Remove "-fopenmp" for g++ version < 4.2
// Usage: make cuda && make run_cuda SPP=500

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "resource/rdtsc.h"

using namespace std;

#define MAX_THREADS 1024
#define MXAA_W 2
#define MXAA_H 2
#define IMG_W 1024
#define IMG_H 768
#define YUKARI 9
// #define YUKARI 11
// #define YUKARI 12

struct Vec
{
    double x, y, z; // position, also color (r,g,b)

    __host__ __device__ Vec(double x_ = 0, double y_ = 0, double z_ = 0)
    {
        x = x_;
        y = y_;
        z = z_;
    }
    __host__ __device__ Vec operator+(const Vec &b) const { return Vec(x + b.x, y + b.y, z + b.z); }
    __host__ __device__ Vec operator-(const Vec &b) const { return Vec(x - b.x, y - b.y, z - b.z); }
    __host__ __device__ Vec operator*(double b) const { return Vec(x * b, y * b, z * b); }
    __host__ __device__ Vec mult(const Vec &b) const { return Vec(x * b.x, y * b.y, z * b.z); }
    __host__ __device__ Vec &norm() { return *this = *this * (1 / sqrt(x * x + y * y + z * z)); }
    __host__ __device__ double dot(const Vec &b) const { return x * b.x + y * b.y + z * b.z; } // cross:
    __host__ __device__ Vec operator%(Vec &b) { return Vec(y * b.z - z * b.y, z * b.x - x * b.z, x * b.y - y * b.x); }
};

struct Ray
{
    Vec o, d;
    __device__ Ray(Vec o_, Vec d_) : o(o_), d(d_) {}
};

enum Refl_t
{
    DIFF,
    SPEC,
    REFR
}; // material types, used in radiance()

struct Sphere
{
    double rad;  // radius
    Vec p, e, c; // position, emission, color
    Refl_t refl; // reflection type (DIFFuse, SPECular, REFRactive)

    __host__ __device__ Sphere(double rad_, Vec p_, Vec e_, Vec c_, Refl_t refl_) : rad(rad_), p(p_), e(e_), c(c_), refl(refl_) {}

    __host__ __device__ double intersect(const Ray &r) const // returns distance, 0 if nohit
    {
        Vec op = p - r.o; // Solve t^2*d.d + 2*t*(o-p).d + (o-p).(o-p)-R^2 = 0
        double t, eps = 1e-4, b = op.dot(r.d), det = b * b - op.dot(op) + rad * rad;
        if (det < 0)
            return 0;
        else
            det = sqrt(det);
        return (t = b - det) > eps ? t : ((t = b + det) > eps ? t : 0);
    }
};

// Cornelbox -------------------------------------------------------------------
// Sphere spheres[] = {
//     // Scene: radius, position, emission, color, material
//     Sphere(1e5, Vec(1e5 + 1, 40.8, 81.6), Vec(), Vec(.75, .25, .25), DIFF),   // Left
//     Sphere(1e5, Vec(-1e5 + 99, 40.8, 81.6), Vec(), Vec(.25, .25, .75), DIFF), // Rght
//     Sphere(1e5, Vec(50, 40.8, 1e5), Vec(), Vec(.75, .75, .75), DIFF),         // Back
//     Sphere(1e5, Vec(50, 40.8, -1e5 + 170), Vec(), Vec(), DIFF),               // Frnt
//     Sphere(1e5, Vec(50, 1e5, 81.6), Vec(), Vec(.75, .75, .75), DIFF),         // Botm
//     Sphere(1e5, Vec(50, -1e5 + 81.6, 81.6), Vec(), Vec(.75, .75, .75), DIFF), // Top
//     Sphere(16.5, Vec(27, 16.5, 47), Vec(), Vec(1, 1, 1) * .999, SPEC),        // Mirr
//     Sphere(16.5, Vec(73, 16.5, 78), Vec(), Vec(1, 1, 1) * .999, REFR),        // Glas
//     Sphere(600, Vec(50, 681.6 - .27, 81.6), Vec(12, 12, 12), Vec(), DIFF)     // Lite
// };

// Sky -------------------------------------------------------------------------
// Vec Cen(50,40.8,-860);
// Sphere spheres[] = {//Scene: radius, position, emission, color, material
//   // center 50 40.8 62
//   // floor 0
//   // back  0

//    Sphere(1600, Vec(1,0,2)*3000, Vec(1,.9,.8)*1.2e1*1.56*2,Vec(), DIFF), // sun
//    Sphere(1560, Vec(1,0,2)*3500,Vec(1,.5,.05)*4.8e1*1.56*2, Vec(),  DIFF), // horizon sun2
// //   Sphere(10000,Cen+Vec(0,0,-200), Vec(0.0627, 0.188, 0.569)*6e-2*8, Vec(.7,.7,1)*.25,  DIFF), // sky
//    Sphere(10000,Cen+Vec(0,0,-200), Vec(0.00063842, 0.02001478, 0.28923243)*6e-2*8, Vec(.7,.7,1)*.25,  DIFF), // sky

//   Sphere(100000, Vec(50, -100000, 0),  Vec(),Vec(.3,.3,.3),DIFF), // grnd
//   Sphere(110000, Vec(50, -110048.5, 0),  Vec(.9,.5,.05)*4,Vec(),DIFF),// horizon brightener
//   Sphere(4e4, Vec(50, -4e4-30, -3000),  Vec(),Vec(.2,.2,.2),DIFF),// mountains
// //  Sphere(3.99e4, Vec(50, -3.99e4+20.045, -3000),  Vec(),Vec(.7,.7,.7),DIFF),// mountains snow

//    Sphere(26.5,Vec(22,26.5,42),   Vec(),Vec(1,1,1)*.596, SPEC), // white Mirr
//    Sphere(13,Vec(75,13,82),   Vec(),Vec(.96,.96,.96)*.96, REFR),// Glas
//   Sphere(22,Vec(87,22,24),   Vec(),Vec(.6,.6,.6)*.696, REFR)    // Glas2
// };

// Nightsky ---------------------------------------------------------------------
Sphere spheres[] = {//Scene: radius, position, emission, color, material
  // center 50 40.8 62
  // floor 0
  // back  0
  //     rad       pos                   emis           col     refl
//  Sphere(1e3,   Vec(1,1,-2)*1e4,    Vec(1,1,1)*5e2,     Vec(), DIFF), // moon
//  Sphere(3e2,   Vec(.6,.2,-2)*1e4,    Vec(1,1,1)*5e3,     Vec(), DIFF), //
//  moon

  Sphere(2.5e3,   Vec(.82,.92,-2)*1e4,    Vec(1,1,1)*.8e2,     Vec(), DIFF), // moon

//  Sphere(2.5e4, Vec(50, 0, 0),     Vec(1,1,1)*1e-3,    Vec(.2,.2,1)*0.0075, DIFF), // sky
//  Sphere(2.5e4, Vec(50, 0, 0),  Vec(0.114, 0.133, 0.212)*1e-2,  Vec(.216,.384,1)*0.0007, DIFF), // sky

  Sphere(2.5e4, Vec(50, 0, 0),  Vec(0.114, 0.133, 0.212)*1e-2,  Vec(.216,.384,1)*0.003, DIFF), // sky

  Sphere(5e0,   Vec(-.2,0.16,-1)*1e4, Vec(1.00, 0.843, 0.698)*1e2,   Vec(), DIFF),  // star
  Sphere(5e0,   Vec(0,  0.18,-1)*1e4, Vec(1.00, 0.851, 0.710)*1e2,  Vec(), DIFF),  // star
  Sphere(5e0,   Vec(.3, 0.15,-1)*1e4, Vec(0.671, 0.780, 1.00)*1e2,   Vec(), DIFF),  // star
  Sphere(3.5e4,   Vec(600,-3.5e4+1, 300), Vec(),   Vec(.6,.8,1)*.01,  REFR),   //pool
  Sphere(5e4,   Vec(-500,-5e4+0, 0),   Vec(),      Vec(1,1,1)*.35,  DIFF),    //hill
  Sphere(16.5,  Vec(27,0,47),         Vec(),              Vec(1,1,1)*.33, DIFF), //hut
  Sphere(7,     Vec(27+8*sqrt(2),0,47+8*sqrt(2)),Vec(),  Vec(1,1,1)*.33,  DIFF), //door
  Sphere(500,   Vec(-1e3,-300,-3e3), Vec(),  Vec(1,1,1)*.351,    DIFF),  //mnt
  Sphere(830,   Vec(0,   -500,-3e3), Vec(),  Vec(1,1,1)*.354,    DIFF),  //mnt
  Sphere(490,  Vec(1e3,  -300,-3e3), Vec(),  Vec(1,1,1)*.352,    DIFF),  //mnt
};

__host__ __device__ double clamp(double x) { return x < 0 ? 0 : x > 1 ? 1 : x; }

__host__ __device__ int toInt(double x) { return int(pow(clamp(x), 1 / 2.2) * 255 + .5); }

__device__ bool intersect(const Ray &r, double &t, int &id, Sphere *spheres)
{
    double d, inf = t = 1e20;
    for (int i = YUKARI; i--;)
        if ((d = spheres[i].intersect(r)) && d < t)
        {
            t = d;
            id = i;
        }
    return t < inf;
}

__device__ Vec radiance(const Ray &r, int depth, hiprandState *state, Sphere* spheres)
{
    Vec result(0.0f, 0.0f, 0.0f);
    Vec weight(1.0f, 1.0f, 1.0f);

    Ray currentRay = r;
    int currentDepth = depth;

    while (true)
    {
        double t;   // distance to intersection
        int id = 0; // id of intersected object

        if (!intersect(currentRay, t, id, spheres)) { // miss, break then return the final color
            break;
        }

        const Sphere &obj = spheres[id]; // the hit object
        Vec x = currentRay.o + currentRay.d * t;
        Vec n = (x - obj.p).norm();
        Vec nl = n.dot(currentRay.d) < 0 ? n : n * -1;
        Vec f = obj.c;

        double p = f.x > f.y && f.x > f.z ? f.x : f.y > f.z ? f.y : f.z; // max refl

        result = result + weight.mult(obj.e);

        if (++currentDepth > 5)
        {
            if (hiprand_uniform(state) < p)
                f = f * (1 / p);
            else {
                // result = result + weight.mult(obj.e); 
                break; // R.R.
            }
        }

        weight = weight.mult(f);

        if (obj.refl == DIFF) // Ideal DIFFUSE reflection
        { 
            double r1 = 2 * M_PI * hiprand_uniform(state);
            double r2 = hiprand_uniform(state);
            double r2s = sqrt(r2);
            Vec w = nl;
            Vec u = ((fabs(w.x) > .1 ? Vec(0, 1) : Vec(1)) % w).norm();
            Vec v = w % u;
            Vec d = (u * cos(r1) * r2s + v * sin(r1) * r2s + w * sqrt(1 - r2)).norm();
            currentRay = Ray(x, d);
            continue;
        }
        else if (obj.refl == SPEC) // Ideal SPECULAR reflection
        { 
            currentRay = Ray(x, currentRay.d - n * 2 * n.dot(currentRay.d));
            continue;
        }
        else // Ideal dielectric REFRACTION
        { 
            Ray reflRay(x, currentRay.d - n * 2 * n.dot(currentRay.d));
            bool into = n.dot(nl) > 0;
            double nc = 1, nt = 1.5;
            double nnt = into ? nc / nt : nt / nc;
            double ddn = currentRay.d.dot(nl), cos2t;

            if ((cos2t = 1 - nnt * nnt * (1 - ddn * ddn)) < 0) // Total internal reflection
            {
                currentRay = reflRay;
                continue;
            }

            Vec tdir = (currentRay.d * nnt - n * ((into ? 1 : -1) * (ddn * nnt + sqrt(cos2t)))).norm();

            double a = nt - nc;
            double b = nt + nc;
            double R0 = a * a / (b * b);
            double c = 1 - (into ? -ddn : tdir.dot(n));
            double Re = R0 + (1 - R0) * c * c * c * c * c;
            double Tr = 1 - Re;
            double P = .25 + .5 * Re;
            double RP = Re / P;
            double TP = Tr / (1 - P);

            if ( hiprand_uniform(state) < P)
            {
                weight = weight * RP;
                currentRay = reflRay;
            }
            else
            {
                weight = weight * TP;
                currentRay = Ray(x, tdir);
            }
        }
    }

    return result;
}

__global__ void render(int samples, Vec *c, Sphere *spheres)
{
  Ray cam(Vec(50, 52, 295.6), Vec(0, -0.042612, -1).norm()); // cam pos, dir
  Vec cx = Vec(IMG_W * .5135 / IMG_H);
  Vec cy = (cx % cam.d).norm() * .5135;
  Vec r(0.0f, 0.0f, 0.0f);

  int x = blockIdx.x * blockDim.x + threadIdx.x; // width
  int y = blockIdx.y * blockDim.y + threadIdx.y; // height
  
  if (!(y < IMG_H && x < IMG_W)) return;
      
  unsigned short Xi = x * x * x + y * y * y; // rand seed
  hiprandState state;
  hiprand_init(Xi, 0, 0, &state); 

  for (int s = 0; s < samples; ++s)
  {
    double r1 = 2 * hiprand_uniform(&state);
    double r2 = 2 * hiprand_uniform(&state);

    double dx = r1 < 1 ? sqrt(r1) - 1 : 1 - sqrt(2 - r1);
    double dy = r2 < 1 ? sqrt(r2) - 1 : 1 - sqrt(2 - r2);

    Vec d = cx * (((0.5 + dx) / 2 + x) / IMG_W - .5) +
            cy * (((0.5 + dy) / 2 + y) / IMG_H - .5) + cam.d;

    r = r + (radiance(Ray(cam.o + d * 140, d.norm()), 0, &state, spheres) * (1.0 / samples));
  }
  
  Vec to_add(0.25 * clamp(r.x), 0.25 * clamp(r.y), 0.25 * clamp(r.z));
  c[(IMG_H-y-1)*IMG_W+x] = c[(IMG_H-y-1)*IMG_W+x] + to_add;

  r = Vec(0.f, 0.f, 0.f);
  for (int s = 0; s < samples; ++s)
  {
    double r1 = 2*hiprand_uniform(&state);
    double r2 = 2*hiprand_uniform(&state);

    double dx = r1 < 1 ? sqrt(r1) - 1 : 1 - sqrt(2 - r1);
    double dy = r2 < 1 ? sqrt(r2) - 1 : 1 - sqrt(2 - r2);

    Vec d = cx * (((0.5 + dx) / 2 + x) / IMG_W - .5) +
            cy * (((1.5 + dy) / 2 + y) / IMG_H - .5) + cam.d;

    r = r + (radiance(Ray(cam.o + d * 140, d.norm()), 0, &state, spheres) * (1.0 / samples));
  }
  to_add = Vec(0.25 * clamp(r.x), 0.25 * clamp(r.y), 0.25 * clamp(r.z));
  c[(IMG_H-y-1)*IMG_W+x] = c[(IMG_H-y-1)*IMG_W+x] + to_add;

  r = Vec(0.f, 0.f, 0.f);
  for (int s = 0; s < samples; ++s)
  {
    double r1 = 2*hiprand_uniform(&state);
    double r2 = 2*hiprand_uniform(&state);

    double dx = r1 < 1 ? sqrt(r1) - 1 : 1 - sqrt(2 - r1);
    double dy = r2 < 1 ? sqrt(r2) - 1 : 1 - sqrt(2 - r2);

    Vec d = cx * (((1.5 + dx) / 2 + x) / IMG_W - .5) +
            cy * (((0.5 + dy) / 2 + y) / IMG_H - .5) + cam.d;

    r = r + (radiance(Ray(cam.o + d * 140, d.norm()), 0, &state, spheres) * (1.0 / samples));
  }
  to_add = Vec(0.25 * clamp(r.x), 0.25 * clamp(r.y), 0.25 * clamp(r.z));
  c[(IMG_H-y-1)*IMG_W+x] = c[(IMG_H-y-1)*IMG_W+x] + to_add;


  r = Vec(0.f, 0.f, 0.f);
  for (int s = 0; s < samples; ++s)
  {
    double r1 = 2*hiprand_uniform(&state);
    double r2 = 2*hiprand_uniform(&state);

    double dx = r1 < 1 ? sqrt(r1) - 1 : 1 - sqrt(2 - r1);
    double dy = r2 < 1 ? sqrt(r2) - 1 : 1 - sqrt(2 - r2);

    Vec d = cx * (((1.5 + dx) / 2 + x) / IMG_W - .5) +
            cy * (((1.5 + dy) / 2 + y) / IMG_H - .5) + cam.d;

    r = r + (radiance(Ray(cam.o + d * 140, d.norm()), 0, &state, spheres) * (1.0 / samples));
  }
  to_add = Vec(0.25 * clamp(r.x), 0.25 * clamp(r.y), 0.25 * clamp(r.z));
  c[(IMG_H-y-1)*IMG_W+x] = c[(IMG_H-y-1)*IMG_W+x] + to_add;
  
  return;
}

int main(int argc, char *argv[])
{
    // Host variables
    int w = IMG_W;
    int h = IMG_H;
    int samps = argc == 2 ? atoi(argv[1]) / MXAA_W / MXAA_H : 1; // # samples
    Vec *result_c = new Vec[w * h];

    // Device variables
    Vec *dev_c;
    hipMalloc((void **)&dev_c, w * h * sizeof(Vec));
    Sphere *dev_spheres;
    hipMalloc((void **)&dev_spheres, sizeof(spheres));
    hipMemcpy(dev_spheres, spheres, sizeof(spheres), hipMemcpyHostToDevice);

    tsc_counter t0, t1;

    RDTSC(t0);

    // Test
    int threadX = 32, threadY = 16;
    // int threadX = 16, threadY = 16;
    // int threadX = 8, threadY = 8;
    // int threadX = 1, threadY = 384;

    dim3 dimGrid(ceil((1.0*w)/threadX), ceil((1.0*h)/threadY), 1);
    dim3 dimBlock(threadX, threadY, 1);
    render<<<dimGrid, dimBlock>>>(samps, dev_c, dev_spheres);

    hipMemcpy(result_c, dev_c, w * h * sizeof(Vec), hipMemcpyDeviceToHost);

    RDTSC(t1);
    printf("\nRendering Time: %lf cycles\n", ((double)COUNTER_DIFF(t1, t0, CYCLES)));

    hipFree(dev_c);
    hipFree(dev_spheres);

    FILE *f = fopen("image.ppm", "w"); // Write image to PPM file.
    fprintf(f, "P3\n%d %d\n%d\n", w, h, 255);
    for (int i = 0; i < w * h; i++)
        fprintf(f, "%d %d %d ", toInt(result_c[i].x), toInt(result_c[i].y), toInt(result_c[i].z));
}
