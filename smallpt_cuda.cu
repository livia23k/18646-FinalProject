#include "hip/hip_runtime.h"
//
// smallpt, a Path Tracer by Kevin Beason, 2008
// Make : g++ -O3 smallpt_serial.cpp -o smallpt_serial
//        Remove "-fopenmp" for g++ version < 4.2
// Usage: time ./smallpt_serial 5000 && xv image.ppm
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "resource/rdtsc.h"

using namespace std;

#define MAX_THREADS 1024
#define MXAA_W 2
#define MXAA_H 2
#define IMG_W 1024
#define IMG_H 768
#define YUKARI 9

struct Vec
{
    double x, y, z; // position, also color (r,g,b)

    __host__ __device__ Vec(double x_ = 0, double y_ = 0, double z_ = 0)
    {
        x = x_;
        y = y_;
        z = z_;
    }
    __host__ __device__ Vec operator+(const Vec &b) const { return Vec(x + b.x, y + b.y, z + b.z); }
    __host__ __device__ Vec operator-(const Vec &b) const { return Vec(x - b.x, y - b.y, z - b.z); }
    __host__ __device__ Vec operator*(double b) const { return Vec(x * b, y * b, z * b); }
    __host__ __device__ Vec mult(const Vec &b) const { return Vec(x * b.x, y * b.y, z * b.z); }
    __host__ __device__ Vec &norm() { return *this = *this * (1 / sqrt(x * x + y * y + z * z)); }
    __host__ __device__ double dot(const Vec &b) const { return x * b.x + y * b.y + z * b.z; } // cross:
    __host__ __device__ Vec operator%(Vec &b) { return Vec(y * b.z - z * b.y, z * b.x - x * b.z, x * b.y - y * b.x); }
};

struct Ray
{
    Vec o, d;
    __device__ Ray(Vec o_, Vec d_) : o(o_), d(d_) {}
};

enum Refl_t
{
    DIFF,
    SPEC,
    REFR
}; // material types, used in radiance()

struct Sphere
{
    double rad;  // radius
    Vec p, e, c; // position, emission, color
    Refl_t refl; // reflection type (DIFFuse, SPECular, REFRactive)

    __host__ __device__ Sphere(double rad_, Vec p_, Vec e_, Vec c_, Refl_t refl_) : rad(rad_), p(p_), e(e_), c(c_), refl(refl_) {}

    __host__ __device__ double intersect(const Ray &r) const // returns distance, 0 if nohit
    {
        Vec op = p - r.o; // Solve t^2*d.d + 2*t*(o-p).d + (o-p).(o-p)-R^2 = 0
        double t, eps = 1e-4, b = op.dot(r.d), det = b * b - op.dot(op) + rad * rad;
        if (det < 0)
            return 0;
        else
            det = sqrt(det);
        return (t = b - det) > eps ? t : ((t = b + det) > eps ? t : 0);
    }
};

// Cornelbox -------------------------------------------------------------------
Sphere spheres[] = {
    // Scene: radius, position, emission, color, material
    Sphere(1e5, Vec(1e5 + 1, 40.8, 81.6), Vec(), Vec(.75, .25, .25), DIFF),   // Left
    Sphere(1e5, Vec(-1e5 + 99, 40.8, 81.6), Vec(), Vec(.25, .25, .75), DIFF), // Rght
    Sphere(1e5, Vec(50, 40.8, 1e5), Vec(), Vec(.75, .75, .75), DIFF),         // Back
    Sphere(1e5, Vec(50, 40.8, -1e5 + 170), Vec(), Vec(), DIFF),               // Frnt
    Sphere(1e5, Vec(50, 1e5, 81.6), Vec(), Vec(.75, .75, .75), DIFF),         // Botm
    Sphere(1e5, Vec(50, -1e5 + 81.6, 81.6), Vec(), Vec(.75, .75, .75), DIFF), // Top
    Sphere(16.5, Vec(27, 16.5, 47), Vec(), Vec(1, 1, 1) * .999, SPEC),        // Mirr
    Sphere(16.5, Vec(73, 16.5, 78), Vec(), Vec(1, 1, 1) * .999, REFR),        // Glas
    Sphere(600, Vec(50, 681.6 - .27, 81.6), Vec(12, 12, 12), Vec(), DIFF)     // Lite
};

// Sky -------------------------------------------------------------------------
// Vec Cen(50,40.8,-860);
// Sphere spheres[] = {//Scene: radius, position, emission, color, material
//   // center 50 40.8 62
//   // floor 0
//   // back  0

//    Sphere(1600, Vec(1,0,2)*3000, Vec(1,.9,.8)*1.2e1*1.56*2,Vec(), DIFF), // sun
//    Sphere(1560, Vec(1,0,2)*3500,Vec(1,.5,.05)*4.8e1*1.56*2, Vec(),  DIFF), // horizon sun2
// //   Sphere(10000,Cen+Vec(0,0,-200), Vec(0.0627, 0.188, 0.569)*6e-2*8, Vec(.7,.7,1)*.25,  DIFF), // sky
//    Sphere(10000,Cen+Vec(0,0,-200), Vec(0.00063842, 0.02001478, 0.28923243)*6e-2*8, Vec(.7,.7,1)*.25,  DIFF), // sky

//   Sphere(100000, Vec(50, -100000, 0),  Vec(),Vec(.3,.3,.3),DIFF), // grnd
//   Sphere(110000, Vec(50, -110048.5, 0),  Vec(.9,.5,.05)*4,Vec(),DIFF),// horizon brightener
//   Sphere(4e4, Vec(50, -4e4-30, -3000),  Vec(),Vec(.2,.2,.2),DIFF),// mountains
// //  Sphere(3.99e4, Vec(50, -3.99e4+20.045, -3000),  Vec(),Vec(.7,.7,.7),DIFF),// mountains snow

//    Sphere(26.5,Vec(22,26.5,42),   Vec(),Vec(1,1,1)*.596, SPEC), // white Mirr
//    Sphere(13,Vec(75,13,82),   Vec(),Vec(.96,.96,.96)*.96, REFR),// Glas
//   Sphere(22,Vec(87,22,24),   Vec(),Vec(.6,.6,.6)*.696, REFR)    // Glas2
// };

// Nightsky ---------------------------------------------------------------------
// Sphere spheres[] = {//Scene: radius, position, emission, color, material
//   // center 50 40.8 62
//   // floor 0
//   // back  0
//   //     rad       pos                   emis           col     refl
// //  Sphere(1e3,   Vec(1,1,-2)*1e4,    Vec(1,1,1)*5e2,     Vec(), DIFF), // moon
// //  Sphere(3e2,   Vec(.6,.2,-2)*1e4,    Vec(1,1,1)*5e3,     Vec(), DIFF), //
// //  moon

//   Sphere(2.5e3,   Vec(.82,.92,-2)*1e4,    Vec(1,1,1)*.8e2,     Vec(), DIFF), // moon

// //  Sphere(2.5e4, Vec(50, 0, 0),     Vec(1,1,1)*1e-3,    Vec(.2,.2,1)*0.0075, DIFF), // sky
// //  Sphere(2.5e4, Vec(50, 0, 0),  Vec(0.114, 0.133, 0.212)*1e-2,  Vec(.216,.384,1)*0.0007, DIFF), // sky

//   Sphere(2.5e4, Vec(50, 0, 0),  Vec(0.114, 0.133, 0.212)*1e-2,  Vec(.216,.384,1)*0.003, DIFF), // sky

//   Sphere(5e0,   Vec(-.2,0.16,-1)*1e4, Vec(1.00, 0.843, 0.698)*1e2,   Vec(), DIFF),  // star
//   Sphere(5e0,   Vec(0,  0.18,-1)*1e4, Vec(1.00, 0.851, 0.710)*1e2,  Vec(), DIFF),  // star
//   Sphere(5e0,   Vec(.3, 0.15,-1)*1e4, Vec(0.671, 0.780, 1.00)*1e2,   Vec(), DIFF),  // star
//   Sphere(3.5e4,   Vec(600,-3.5e4+1, 300), Vec(),   Vec(.6,.8,1)*.01,  REFR),   //pool
//   Sphere(5e4,   Vec(-500,-5e4+0, 0),   Vec(),      Vec(1,1,1)*.35,  DIFF),    //hill
//   Sphere(16.5,  Vec(27,0,47),         Vec(),              Vec(1,1,1)*.33, DIFF), //hut
//   Sphere(7,     Vec(27+8*sqrt(2),0,47+8*sqrt(2)),Vec(),  Vec(1,1,1)*.33,  DIFF), //door
//   Sphere(500,   Vec(-1e3,-300,-3e3), Vec(),  Vec(1,1,1)*.351,    DIFF),  //mnt
//   Sphere(830,   Vec(0,   -500,-3e3), Vec(),  Vec(1,1,1)*.354,    DIFF),  //mnt
//   Sphere(490,  Vec(1e3,  -300,-3e3), Vec(),  Vec(1,1,1)*.352,    DIFF),  //mnt
// };

__host__ __device__ double clamp(double x) { return x < 0 ? 0 : x > 1 ? 1
                                                                      : x; }

__host__ __device__ int toInt(double x) { return int(pow(clamp(x), 1 / 2.2) * 255 + .5); }

__device__ bool intersect(const Ray &r, double &t, int &id, Sphere *spheres)
{
    double d, inf = t = 1e20;
    for (int i = YUKARI; i--;)
        if ((d = spheres[i].intersect(r)) && d < t)
        {
            t = d;
            id = i;
        }
    return t < inf;
}

// __device__ Vec radiance(const Ray &r, int depth, hiprandState *state, Sphere *spheres)
// {
//     double t;   // distance to intersection
//     int id = 0; // id of intersected object

//     if (!intersect(r, t, id, spheres))
//         return Vec(1.0f, 0.0f, 1.0f); // if miss, return black

//     const Sphere &obj = spheres[id]; // the hit object
//     Vec x = r.o + r.d * t;
//     Vec n = (x - obj.p).norm();
//     Vec nl = n.dot(r.d) < 0 ? n : n * -1;
//     Vec f = obj.c;

//     double p = f.x > f.y && f.x > f.z ? f.x : f.y > f.z ? f.y
//                                                         : f.z; // max refl

//     if (++depth > 5)
//         if (hiprand_uniform(state) < p)
//             f = f * (1 / p);
//         else
//             return obj.e; // R.R.

//     if (obj.refl == DIFF) // Ideal DIFFUSE reflection
//     {

//         double r1 = 2 * M_PI * hiprand_uniform(state), r2 = hiprand_uniform(state), r2s = sqrt(r2);
//         Vec w = nl;
//         Vec u = ((fabs(w.x) > .1 ? Vec(0, 1) : Vec(1)) % w).norm();
//         Vec v = w % u;
//         Vec d = (u * cos(r1) * r2s + v * sin(r1) * r2s + w * sqrt(1 - r2)).norm();
//         return obj.e + f.mult(radiance(Ray(x, d), depth, state, spheres));
//     }
//     else if (obj.refl == SPEC) // Ideal SPECULAR reflection
//     {
//         return obj.e + f.mult(radiance(Ray(x, r.d - n * 2 * n.dot(r.d)), depth, state, spheres));
//     }

//     Ray reflRay(x, r.d - n * 2 * n.dot(r.d)); // Ideal dielectric REFRACTION

//     bool into = n.dot(nl) > 0; // Ray from outside going in?

//     double nc = 1, nt = 1.5;
//     double nnt = into ? nc / nt : nt / nc;
//     double ddn = r.d.dot(nl), cos2t;

//     if ((cos2t = 1 - nnt * nnt * (1 - ddn * ddn)) < 0) // Total internal reflection
//         return obj.e + f.mult(radiance(reflRay, depth, state, spheres));

//     Vec tdir = (r.d * nnt - n * ((into ? 1 : -1) * (ddn * nnt + sqrt(cos2t)))).norm();

//     double a = nt - nc;
//     double b = nt + nc;
//     double R0 = a * a / (b * b);
//     double c = 1 - (into ? -ddn : tdir.dot(n));
//     double Re = R0 + (1 - R0) * c * c * c * c * c;
//     double Tr = 1 - Re;
//     double P = .25 + .5 * Re;
//     double RP = Re / P;
//     double TP = Tr / (1 - P);

//     return obj.e + f.mult(depth > 2 ? (hiprand_uniform(state) < P ? // Russian roulette
//                                            radiance(reflRay, depth, state, spheres) * RP
//                                                                  : radiance(Ray(x, tdir), depth, state, spheres) * TP)
//                                     : radiance(reflRay, depth, state, spheres) * Re + radiance(Ray(x, tdir), depth, state, spheres) * Tr);
// }

__device__ Vec radiance(const Ray _r, int _depth, hiprandState *state, Sphere *spheres)
{
    double t;    // distance to intersection
    int id = 0; // id of intersected object
    Ray r = _r;
    int depth = _depth;
    Vec cl(0, 0, 0); // accumulated color
    Vec cf(1, 1, 1); // accumulated reflectance
    while (1)
    {
        if (!intersect(r, t, id, spheres))
            return cl;                   // if miss, return black
        const Sphere &obj = spheres[id]; // the hit object
        Vec x = r.o + r.d * t, n = (x - obj.p).norm(), nl = n.dot(r.d) < 0 ? n : n * -1, f = obj.c;
        float p = f.x > f.y && f.x > f.z ? f.x : f.y > f.z ? f.y
                                                           : f.z; // max refl
        cl = cl + cf.mult(obj.e);
        if (++depth > 5)
            if (hiprand_uniform(state) < p)
                f = f * (1 / p);
            else
                return cl; // R.R.
        cf = cf.mult(f);
        if (obj.refl == DIFF)
        { // Ideal DIFFUSE reflection
            double r1 = 2 * M_PI * hiprand_uniform(state), r2 = hiprand_uniform(state), r2s = sqrt(r2);
            Vec w = nl, u = ((fabs(w.x) > .1 ? Vec(0, 1) : Vec(1)) % w).norm(), v = w % u;
            Vec d = (u * cos(r1) * r2s + v * sin(r1) * r2s + w * sqrt(1 - r2)).norm();
            // return obj.e + f.mult(radiance(Ray(x,d),depth,Xi));
            r = Ray(x, d);
            continue;
        }
        else if (obj.refl == SPEC)
        { // Ideal SPECULAR reflection
            // return obj.e + f.mult(radiance(Ray(x,r.d-n*2*n.dot(r.d)),depth,Xi));
            r = Ray(x, r.d - n * 2 * n.dot(r.d));
            continue;
        }
        Ray reflRay(x, r.d - n * 2 * n.dot(r.d)); // Ideal dielectric REFRACTION
        bool into = n.dot(nl) > 0;                // Ray from outside going in?
        int nc = 1;
        float nt = 1.5, nnt = into ? nc / nt : nt / nc, ddn = r.d.dot(nl), cos2t;
        if ((cos2t = 1 - nnt * nnt * (1 - ddn * ddn)) < 0)
        { // Total internal reflection
            // return obj.e + f.mult(radiance(reflRay,depth,Xi));
            r = reflRay;
            continue;
        }
        Vec tdir = (r.d * nnt - n * ((into ? 1 : -1) * (ddn * nnt + sqrt(cos2t)))).norm();
        float a = nt - nc, b = nt + nc, R0 = a * a / (b * b), c = 1 - (into ? -ddn : tdir.dot(n));
        float Re = R0 + (1 - R0) * c * c * c * c * c, Tr = 1 - Re, P = .25f + .5f * Re, RP = Re / P, TP = Tr / (1 - P);
        // return obj.e + f.mult(erand48(Xi)<P ?
        //                       radiance(reflRay,    depth,Xi)*RP:
        //                       radiance(Ray(x,tdir),depth,Xi)*TP);
        if (hiprand_uniform(state) < P)
        {
            cf = cf * RP;
            r = reflRay;
        }
        else
        {
            cf = cf * TP;
            r = Ray(x, tdir);
        }
        continue;
    }
}

__global__ void render(int samples, Vec *c, Sphere *spheres)
{
    Ray cam(Vec(50, 52, 295.6), Vec(0, -0.042612, -1).norm()); // cam pos, dir
    Vec cx = Vec(IMG_W * .5135 / IMG_H);
    Vec cy = (cx % cam.d).norm() * .5135;
    Vec r(0.0f, 0.0f, 0.0f);

    int x = blockIdx.x * blockDim.x + threadIdx.x; // horizontal
    int y = blockIdx.y * blockDim.y + threadIdx.y; // vertical
    // int id = blockIdx.x * blockDim.x + threadIdx.x;
    // int y = blockIdx.x;
    // int x = threadIdx.x; // vertical

    unsigned short Xi = y * y * y;

    ////test screen space
    // Vec t(x / 1024.0f , y / 768.0f, 1.0f);
    // atomicAddVec(&c[(IMG_H - y - 1) * IMG_W + x], t);
    // return;
    ////PASS

    if (!(y < IMG_H && x < IMG_W))
        return;
        
    hiprandState state;
    //hipMalloc(&state, sizeof(hiprandState));
    hiprand_init(Xi, 0, 0, &state); // the seed for the same thread will always
                                                                    // be the same, but distinct from other threads



  // int x = blockIdx.x * blockDim.x + threadIdx.x;
  // int y = blockIdx.y * blockDim.y + threadIdx.y;
  ////test single MXAA
  // if (!(y<IMG_H && x<IMG_W)) return;

//   for (int s = 0; s < samples; ++s)
//   {
//     double r1 = 2*hiprand_uniform(&state);
//     double r2 = 2*hiprand_uniform(&state);

//     double dx = r1 < 1 ? sqrt(r1) - 1 : 1 - sqrt(2 - r1);
//     double dy = r2 < 1 ? sqrt(r2) - 1 : 1 - sqrt(2 - r2);

//     Vec d = cx * (((1 + dx) / 2 + x) / IMG_W - .5) +
//             cy * (((1 + dy) / 2 + y) / IMG_H - .5) + cam.d;

//     r = r + (radiance(Ray(cam.o + d * 140, d.norm()), 0, &state, spheres) * (1.0 / samples));
//   }
//   Vec to_add(0.25 * clamp(r.x), 0.25 * clamp(r.y), 0.25 * clamp(r.z));
//   c[(IMG_H-y-1)*IMG_W+x] = c[(IMG_H-y-1)*IMG_W+x] + to_add*4;
//   return;


  for (int s = 0; s < samples; ++s)
  {
    double r1 = 2*hiprand_uniform(&state);
    double r2 = 2*hiprand_uniform(&state);

    double dx = r1 < 1 ? sqrt(r1) - 1 : 1 - sqrt(2 - r1);
    double dy = r2 < 1 ? sqrt(r2) - 1 : 1 - sqrt(2 - r2);

    Vec d = cx * (((0.5 + dx) / 2 + x) / IMG_W - .5) +
            cy * (((0.5 + dy) / 2 + y) / IMG_H - .5) + cam.d;

    r = r + (radiance(Ray(cam.o + d * 140, d.norm()), 0, &state, spheres) * (1.0 / samples));
  }
  Vec to_add(0.25 * clamp(r.x), 0.25 * clamp(r.y), 0.25 * clamp(r.z));
  c[(IMG_H-y-1)*IMG_W+x] = c[(IMG_H-y-1)*IMG_W+x] + to_add;


  // if (!(y<IMG_H && x<IMG_W)) return;
  r = Vec(0.f, 0.f, 0.f);
  for (int s = 0; s < samples; ++s)
  {
    double r1 = 2*hiprand_uniform(&state);
    double r2 = 2*hiprand_uniform(&state);

    double dx = r1 < 1 ? sqrt(r1) - 1 : 1 - sqrt(2 - r1);
    double dy = r2 < 1 ? sqrt(r2) - 1 : 1 - sqrt(2 - r2);

    Vec d = cx * (((0.5 + dx) / 2 + x) / IMG_W - .5) +
            cy * (((1.5 + dy) / 2 + y) / IMG_H - .5) + cam.d;

    r = r + (radiance(Ray(cam.o + d * 140, d.norm()), 0, &state, spheres) * (1.0 / samples));
  }
  to_add = Vec(0.25 * clamp(r.x), 0.25 * clamp(r.y), 0.25 * clamp(r.z));
  c[(IMG_H-y-1)*IMG_W+x] = c[(IMG_H-y-1)*IMG_W+x] + to_add;


    // if (!(y<IMG_H && x<IMG_W)) return;
  r = Vec(0.f, 0.f, 0.f);
  for (int s = 0; s < samples; ++s)
  {
    double r1 = 2*hiprand_uniform(&state);
    double r2 = 2*hiprand_uniform(&state);

    double dx = r1 < 1 ? sqrt(r1) - 1 : 1 - sqrt(2 - r1);
    double dy = r2 < 1 ? sqrt(r2) - 1 : 1 - sqrt(2 - r2);

    Vec d = cx * (((1.5 + dx) / 2 + x) / IMG_W - .5) +
            cy * (((0.5 + dy) / 2 + y) / IMG_H - .5) + cam.d;

    r = r + (radiance(Ray(cam.o + d * 140, d.norm()), 0, &state, spheres) * (1.0 / samples));
  }
  to_add = Vec(0.25 * clamp(r.x), 0.25 * clamp(r.y), 0.25 * clamp(r.z));
  c[(IMG_H-y-1)*IMG_W+x] = c[(IMG_H-y-1)*IMG_W+x] + to_add;


    // if (!(y<IMG_H && x<IMG_W)) return;
  r = Vec(0.f, 0.f, 0.f);
  for (int s = 0; s < samples; ++s)
  {
    double r1 = 2*hiprand_uniform(&state);
    double r2 = 2*hiprand_uniform(&state);

    double dx = r1 < 1 ? sqrt(r1) - 1 : 1 - sqrt(2 - r1);
    double dy = r2 < 1 ? sqrt(r2) - 1 : 1 - sqrt(2 - r2);

    Vec d = cx * (((1.5 + dx) / 2 + x) / IMG_W - .5) +
            cy * (((1.5 + dy) / 2 + y) / IMG_H - .5) + cam.d;

    r = r + (radiance(Ray(cam.o + d * 140, d.norm()), 0, &state, spheres) * (1.0 / samples));
  }
  to_add = Vec(0.25 * clamp(r.x), 0.25 * clamp(r.y), 0.25 * clamp(r.z));
  c[(IMG_H-y-1)*IMG_W+x] = c[(IMG_H-y-1)*IMG_W+x] + to_add;
  
  return;



    // for (int sy = 0, i = (IMG_H - y - 1) * IMG_W + x; sy < 2; sy++)
    // {                                             // MXAA rows
    //     for (int sx = 0; sx < 2; sx++, r = Vec()) // MXAA cols
    //         for (int s = 0; s < samples; ++s)
    //         { // Ray samples per MXAA sample

    //             double r1 = 2 * hiprand_uniform(&state);
    //             double r2 = 2 * hiprand_uniform(&state);

    //             double dx = r1 < 1 ? sqrt(r1) - 1 : 1 - sqrt(2 - r1);
    //             double dy = r2 < 1 ? sqrt(r2) - 1 : 1 - sqrt(2 - r2);

    //             Vec d = cx * (((sx + .5 + dx) / 2 + x) / IMG_W - .5) +
    //                     cy * (((sy + .5 + dy) / 2 + y) / IMG_H - .5) + cam.d;

    //             r = r + radiance(Ray(cam.o + d * 140, d.norm()), 0, &state, spheres) * (1.0f / samples);
    //         }

    //     // c[i] = c[i] + Vec(clamp(r.x), clamp(r.y), clamp(r.z)) * .25;
    //     Vec to_add(0.25 * clamp(r.x), 0.25 * clamp(r.y), 0.25 * clamp(r.z));
    //     c[i] = c[i] + to_add;
    // }
}

int main(int argc, char *argv[])
{
    // Host variables
    int w = IMG_W;
    int h = IMG_H;
    int samps = argc == 2 ? atoi(argv[1]) / MXAA_W / MXAA_H : 1; // # samples
    Vec *result_c = new Vec[w * h];

    // Device variables
    Vec *dev_c;
    hipMalloc((void **)&dev_c, w * h * sizeof(Vec));
    Sphere *dev_spheres;
    hipMalloc((void **)&dev_spheres, sizeof(spheres));
    hipMemcpy(dev_spheres, spheres, sizeof(spheres), hipMemcpyHostToDevice);

    tsc_counter t0, t1;

    RDTSC(t0);

    // render<<<768, 1024>>>(samps, dev_c, dev_spheres);

    dim3 dimGrid(ceil((1.0*w)/32), ceil((1.0*h)/16), 1);
    dim3 dimBlock(32, 16, 1);
    render<<<dimGrid, dimBlock>>>(samps, dev_c, dev_spheres);

    hipMemcpy(result_c, dev_c, w * h * sizeof(Vec), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_spheres);
    // */

    RDTSC(t1);
    printf("\nRendering Time: %lf cycles\n", ((double)COUNTER_DIFF(t1, t0, CYCLES)));

    FILE *f = fopen("image.ppm", "w"); // Write image to PPM file.
    fprintf(f, "P3\n%d %d\n%d\n", w, h, 255);
    for (int i = 0; i < w * h; i++)
        fprintf(f, "%d %d %d ", toInt(result_c[i].x), toInt(result_c[i].y), toInt(result_c[i].z));
}
